#include "hip/hip_runtime.h"
/*
 * Authors: Tomas Kocisky
 *
 * Faster component-wise "part 2" of LSTM.
 */
#include "utils.h"

template<class F>
struct lstmpart2Output_functor
{
  inline __host__ __device__ F s(const F& f) const {
    return 1./(1. + exp(-f));
  }
  inline __host__ __device__ F ds(const F& sf) const {
    return (1. - sf) * sf;
  }
  inline __host__ __device__ F t(const F& f) const {
    return tanh(f);
  }
  inline __host__ __device__ F dt(const F& tf) const {
    return 1. - tf * tf;
  }

  __host__ __device__ F operator()(const F& pc, const F& fg, const F& ig,
                                   const F& i, const F& og, F& nc, F& nh) const
  {
    nc = s(fg) * pc + s(ig) * t(i);
    nh = s(og) * t(nc);
    return 0;
  }
};

template<class F>
struct lstmpart2GradInput_functor
{
  inline __host__ __device__ F s(const F& f) const {
    return 1./(1. + exp(-f));
  }
  inline __host__ __device__ F ds(const F& sf) const {
    return (1. - sf) * sf;
  }
  inline __host__ __device__ F t(const F& f) const {
    return tanh(f);
  }
  inline __host__ __device__ F dt(const F& tf) const {
    return 1. - tf * tf;
  }

  __host__ __device__ F operator()(
      F& pc, F& fg, F& ig, F& i, F& og,
      const F& gonc, const F& gonh
      ) const
  {
    float sfg = s(fg);
    float sig = s(ig);
    float sog = s(og);
    float ti = t(i);
    float tnc = t(sfg * pc + sig * ti);
    og = gonh * tnc * ds(sog);
    float gnc = gonc + gonh * sog * dt(tnc);
    fg = gnc * pc * ds(sfg);
    pc = gnc * sfg;
    ig = gnc * ti * ds(sig);
    i = gnc * sig * dt(ti);
    return 0;
  }
};

// from https://thrust.github.io/doc/classthrust_1_1iterator__adaptor.html
#include <thrust/iterator/iterator_adaptor.h>
template<typename Iterator>
class skip_iterator
  : public thrust::iterator_adaptor< skip_iterator<Iterator>, Iterator >
{
 public:
  typedef thrust::iterator_adaptor<skip_iterator<Iterator>, Iterator > super_t;

  __host__ __device__
  skip_iterator(const Iterator &x, int n) : super_t(x), begin(x), n(n) {}
  friend class thrust::iterator_core_access;
 private:
  unsigned int n;
  const Iterator begin;

  __host__ __device__
  typename super_t::reference dereference() const {
    return *(begin + (this->base() - begin) * n);
  }
};

template<typename Iterator>
skip_iterator<Iterator> make_skip_iterator(Iterator it, int n)
{ return skip_iterator<Iterator>(it,n); }

template<typename SevenaryFunction>
struct tuple7_transform_functor
{
  SevenaryFunction f;

  __host__ __device__ tuple7_transform_functor(SevenaryFunction f) : f(f) {}

  template<typename Tuple>
  inline __host__ __device__ void operator()(Tuple t) {
    f(
        thrust::get<0>(t),
        thrust::get<1>(t),
        thrust::get<2>(t),
        thrust::get<3>(t),
        thrust::get<4>(t),
        thrust::get<5>(t),
        thrust::get<6>(t)
      );
  }
};

static int oxnn_LSTM12Part2_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *prev_c = (THCudaTensor*)luaT_checkudata(L, 1,
                                                        "torch.CudaTensor");
  THCudaTensor *raw_gates = (THCudaTensor*)luaT_checkudata(L, 2,
                                                        "torch.CudaTensor");
  THCudaTensor *next_c = (THCudaTensor*)luaT_checkudata(L, 3,
                                                        "torch.CudaTensor");
  THCudaTensor *next_h = (THCudaTensor*)luaT_checkudata(L, 4,
                                                        "torch.CudaTensor");

  long size = THCudaTensor_nElement(state, prev_c);
  long size_g = THCudaTensor_nElement(state, raw_gates);
  long size_nc = THCudaTensor_nElement(state, next_c);
  long size_nh = THCudaTensor_nElement(state, next_h);

  assert(THCudaTensor_isContiguous(state, prev_c));
  assert(THCudaTensor_isContiguous(state, raw_gates));
  assert(THCudaTensor_isContiguous(state, next_c));
  assert(THCudaTensor_isContiguous(state, next_h));
  assert(size * 4 == size_g);
  assert(size == size_nc);
  assert(size == size_nh);

  typedef tuple7_transform_functor<lstmpart2Output_functor<float> >
    SevenaryTransformFunctor;

  thrust::device_ptr<float> prev_c_data(THCudaTensor_data(state, prev_c));
  thrust::device_ptr<float> raw_gates_data(THCudaTensor_data(state, raw_gates));
  thrust::device_ptr<float> next_c_data(THCudaTensor_data(state, next_c));
  thrust::device_ptr<float> next_h_data(THCudaTensor_data(state, next_h));
  thrust::for_each(
                   thrust::make_zip_iterator(thrust::make_tuple(
                       prev_c_data,
                       make_skip_iterator(raw_gates_data+0, 4),
                       make_skip_iterator(raw_gates_data+1, 4),
                       make_skip_iterator(raw_gates_data+2, 4),
                       make_skip_iterator(raw_gates_data+3, 4),
                       next_c_data,
                       next_h_data
                       )),
                   thrust::make_zip_iterator(thrust::make_tuple(
                       prev_c_data+size,
                       make_skip_iterator(raw_gates_data+0, 4),
                       make_skip_iterator(raw_gates_data+1, 4),
                       make_skip_iterator(raw_gates_data+2, 4),
                       make_skip_iterator(raw_gates_data+3, 4),
                       next_c_data,
                       next_h_data
                       )),
                   SevenaryTransformFunctor(lstmpart2Output_functor<float>())
                  );
  return 1;
}


static int oxnn_LSTM12Part2_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *prev_c = (THCudaTensor*)luaT_checkudata(L, 1,
                                                        "torch.CudaTensor");
  THCudaTensor *raw_gates = (THCudaTensor*)luaT_checkudata(L, 2,
                                                        "torch.CudaTensor");
  THCudaTensor *GOnext_c = (THCudaTensor*)luaT_checkudata(L, 3,
                                                        "torch.CudaTensor");
  THCudaTensor *GOnext_h = (THCudaTensor*)luaT_checkudata(L, 4,
                                                        "torch.CudaTensor");

  long size = THCudaTensor_nElement(state, prev_c);
  long size_g = THCudaTensor_nElement(state, raw_gates);
  long size_nc = THCudaTensor_nElement(state, GOnext_c);
  long size_nh = THCudaTensor_nElement(state, GOnext_h);

  assert(THCudaTensor_isContiguous(state, prev_c));
  assert(THCudaTensor_isContiguous(state, raw_gates));
  assert(THCudaTensor_isContiguous(state, GOnext_c));
  assert(THCudaTensor_isContiguous(state, GOnext_h));
  assert(size * 4 == size_g);
  assert(size == size_nc);
  assert(size == size_nh);


  typedef tuple7_transform_functor<lstmpart2GradInput_functor<float> >
    SevenaryTransformFunctor;

  thrust::device_ptr<float> prev_c_data(THCudaTensor_data(state, prev_c));
  thrust::device_ptr<float> raw_gates_data(THCudaTensor_data(state, raw_gates));
  thrust::device_ptr<float> GOnext_c_data(THCudaTensor_data(state, GOnext_c));
  thrust::device_ptr<float> GOnext_h_data(THCudaTensor_data(state, GOnext_h));
  thrust::for_each(
                   thrust::make_zip_iterator(thrust::make_tuple(
                       prev_c_data,
                       make_skip_iterator(raw_gates_data+0, 4),
                       make_skip_iterator(raw_gates_data+1, 4),
                       make_skip_iterator(raw_gates_data+2, 4),
                       make_skip_iterator(raw_gates_data+3, 4),
                       GOnext_c_data,
                       GOnext_h_data
                       )),
                   thrust::make_zip_iterator(thrust::make_tuple(
                       prev_c_data+size,
                       make_skip_iterator(raw_gates_data+0, 4),
                       make_skip_iterator(raw_gates_data+1, 4),
                       make_skip_iterator(raw_gates_data+2, 4),
                       make_skip_iterator(raw_gates_data+3, 4),
                       GOnext_c_data,
                       GOnext_h_data
                       )),
                   SevenaryTransformFunctor(
                     lstmpart2GradInput_functor<float>())
                  );
  return 1;
}

static const struct luaL_Reg oxnn_LSTM12Part2__ [] = {
  {"LSTM12Part2_updateOutput", oxnn_LSTM12Part2_updateOutput},
  {"LSTM12Part2_updateGradInput", oxnn_LSTM12Part2_updateGradInput},
  {NULL, NULL}
};

static void oxnn_LSTM12Part2_init(lua_State *L) {
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, oxnn_LSTM12Part2__, "oxnn");
  lua_pop(L,1);
}
